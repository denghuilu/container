#include "hip/hip_runtime.h"
#include "../memory_op.h"

#include <complex>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#define THREADS_PER_BLOCK 256

namespace container {
namespace op {

template <typename T_out, typename T_in>
__global__ void cast_memory(
        T_out* out,
        const T_in* in,
        const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    out[idx] = static_cast<T_out>(in[idx]);
}

template <typename T_out, typename T_in>
__global__ void cast_memory(
        std::complex<T_out>* out,
        const std::complex<T_in>* in,
        const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    auto* _out = reinterpret_cast<thrust::complex<T_out>*>(out);
    const auto* _in = reinterpret_cast<const thrust::complex<T_in>*>(in);
    _out[idx] = static_cast<thrust::complex<T_out>>(_in[idx]);
}

template <typename T>
void resize_memory_op<T, container::DEVICE_GPU>::operator()(
    const container::DEVICE_GPU* dev,
    T*& arr,
    const size_t size,
    const char* record_in)
{
  if (arr != nullptr) {
    delete_memory_op<T, container::DEVICE_GPU>()(dev, arr);
  }
  hipMalloc((void **)&arr, sizeof(T) * size);
}

template <typename T>
void set_memory_op<T, container::DEVICE_GPU>::operator()(
    T* arr,
    const int var, 
    const size_t size) 
{
  hipMemset(arr, var, sizeof(T) * size);
}

template <typename T>
void synchronize_memory_op<T, container::DEVICE_CPU, container::DEVICE_GPU>::operator()(
    T* arr_out,
    const T* arr_in,
    const size_t size) 
{
  hipMemcpy(arr_out, arr_in, sizeof(T) * size, hipMemcpyDeviceToHost);
}

template <typename T>
void synchronize_memory_op<T, container::DEVICE_GPU, container::DEVICE_CPU>::operator()(
    T* arr_out,
    const T* arr_in,
    const size_t size) 
{
  hipMemcpy(arr_out, arr_in, sizeof(T) * size, hipMemcpyHostToDevice);
}

template <typename T>
void synchronize_memory_op<T, container::DEVICE_GPU, container::DEVICE_GPU>::operator()(
    T* arr_out,
    const T* arr_in,
    const size_t size) 
{
  hipMemcpy(arr_out, arr_in, sizeof(T) * size, hipMemcpyDeviceToDevice);
}

template <typename T_out, typename T_in>
struct cast_memory_op<T_out, T_in, container::DEVICE_GPU, container::DEVICE_GPU> {
    void operator()(T_out* arr_out,
                    const T_in* arr_in,
                    const size_t size) {
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        cast_memory<<<block, THREADS_PER_BLOCK>>>(arr_out, arr_in, size);
    }
};

template <typename T_out, typename T_in>
struct cast_memory_op<T_out, T_in, container::DEVICE_GPU, container::DEVICE_CPU> {
    void operator()(T_out* arr_out,
                    const T_in* arr_in,
                    const size_t size) {
        T_in * arr = nullptr;
        hipMalloc((void **)&arr, sizeof(T_in) * size);
        hipMemcpy(arr, arr_in, sizeof(T_in) * size, hipMemcpyHostToDevice);
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        cast_memory<<<block, THREADS_PER_BLOCK>>>(arr_out, arr, size);
        hipFree(arr);
    }
};

template <typename T_out, typename T_in>
struct cast_memory_op<T_out, T_in, container::DEVICE_CPU, container::DEVICE_GPU> {
    void operator()(T_out* arr_out,
                    const T_in* arr_in,
                    const size_t size) {
        auto * arr = (T_in*) malloc(sizeof(T_in) * size);
        hipMemcpy(arr, arr_in, sizeof(T_in) * size, hipMemcpyDeviceToHost);
        for (int ii = 0; ii < size; ii++) {
            arr_out[ii] = static_cast<T_out>(arr[ii]);
        }
        free(arr);
    }
};

template <typename T>
void delete_memory_op<T, container::DEVICE_GPU>::operator() (
    const container::DEVICE_GPU* dev,
    T* arr)
{
  hipFree(arr);
}

template struct resize_memory_op<int, container::DEVICE_GPU>;
template struct resize_memory_op<int64_t, container::DEVICE_GPU>;
template struct resize_memory_op<float, container::DEVICE_GPU>;
template struct resize_memory_op<double, container::DEVICE_GPU>;
template struct resize_memory_op<std::complex<float>, container::DEVICE_GPU>;
template struct resize_memory_op<std::complex<double>, container::DEVICE_GPU>;

template struct set_memory_op<int, container::DEVICE_GPU>;
template struct set_memory_op<int64_t , container::DEVICE_GPU>;
template struct set_memory_op<float, container::DEVICE_GPU>;
template struct set_memory_op<double, container::DEVICE_GPU>;
template struct set_memory_op<std::complex<float>, container::DEVICE_GPU>;
template struct set_memory_op<std::complex<double>, container::DEVICE_GPU>;

template struct synchronize_memory_op<int, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<int, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<int, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<int64_t, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<int64_t, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<int64_t, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<float, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<float, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<float, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<double, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<double, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<double, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<float>, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<double>, container::DEVICE_GPU, container::DEVICE_GPU>;

template struct cast_memory_op<float, float, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<double, double, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<float, double, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<double, float, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, std::complex<float>, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, std::complex<double>, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, std::complex<double>, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, std::complex<float>, container::DEVICE_GPU, container::DEVICE_GPU>;
template struct cast_memory_op<float, float, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<double, double, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<float, double, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<double, float, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>, std::complex<float>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>, std::complex<double>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>, std::complex<double>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>, std::complex<float>, container::DEVICE_GPU, container::DEVICE_CPU>;
template struct cast_memory_op<float, float, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<double, double, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<float, double, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<double, float, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, std::complex<float>, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, std::complex<double>, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, std::complex<double>, container::DEVICE_CPU, container::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, std::complex<float>, container::DEVICE_CPU, container::DEVICE_GPU>;

template struct delete_memory_op<int, container::DEVICE_GPU>;
template struct delete_memory_op<int64_t, container::DEVICE_GPU>;
template struct delete_memory_op<float, container::DEVICE_GPU>;
template struct delete_memory_op<double, container::DEVICE_GPU>;
template struct delete_memory_op<std::complex<float>, container::DEVICE_GPU>;
template struct delete_memory_op<std::complex<double>, container::DEVICE_GPU>;

} // end of namespace container
} // end of namespace op